#include "hip/hip_runtime.h"
/*sigmoid function
* Author    : Huang Daoji
* StudentID : 1600017857
* Date      : 2018-04-16
*/

// header files
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M_PI_2        1.57079632679489661923	/* pi/2 */
#define M_PI_2_INV    (1.0/M_PI_2)
#define M_2_SQRTPI    1.12837916709551257390    /* 2/sqrt(pi) */
#define ERF_COEF      (1.0/M_2_SQRTPI)
#define threhold      100000000


void verification(){
    double ans = 0.0;
    for(double i = 1.0; i <= 100000000.0; i += 1.0){
        ans += 1 / (1 + exp(-i));
    }
    printf("result on cpu: %10.10f. \n", ans);
}

// print some basic parameters
void printDeviceProp(const hipDeviceProp_t &prop) {
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

// find a device, and quit
bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printDeviceProp(prop);
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}


// kernel function here
__global__ static void calc(double* gpuans) {
	__shared__ double tmp[100];
	int bid = blockIdx.x * 100 + blockIdx.y;
	int tid = threadIdx.x * 10 + threadIdx.y;

	double idx = bid * 10000 + tid * 100;
	double ans = 0.0;
	if (idx < threhold) {
		for (double i = 1.0; i <= 100.0; i += 1.0) {
			ans += 1 / (1 + exp(-idx - i));
		}
	}
	else {
		for (double i = 1.0; i <= 100.0; i += 1.0) {
			ans += M_PI_2_INV * atan(M_PI_2 * (i + idx));
		}
	}
	tmp[tid] = ans;
    __syncthreads();
    /*
	int offset = 1, mask = 1;
	while (offset < 100) {
		if ((tid & mask) == 0 && tid + mask < 100) {
			tmp[tid] += tmp[tid + offset];
		}
		offset = offset << 1;
		mask = mask + offset;
		__syncthreads();
	}*/
    int i = 50;
    while(i != 0){
        if(tid < i){
            tmp[tid] = tmp[tid + i] + tmp[tid];
        }
        __syncthreads();
        i /= 2;
    }
	if (tid == 0) {
        /*
        double res = 0.0;
        for(int i = 0; i < 100; i += 1){
            res += tmp[i];
        }*/
		gpuans[bid] = tmp[0];
	}
}

int main() {
	if (!InitCUDA()) {
		return 0;
	}

	// warmup
	double* warmup = (double*)malloc(sizeof(double) * 1024 * 1024);
	double* gpuwarmup;
	hipMalloc((void**)&gpuwarmup, sizeof(double) * 1024 * 1024);
	hipMemcpy(gpuwarmup, warmup, sizeof(double) * 1024 * 1024, hipMemcpyHostToDevice);

	//
	dim3 dimBlock(10, 10);
	dim3 dimGrid(100, 100);

	clock_t start, stop;
	start = clock();
	double* ans = (double*)calloc(10000, sizeof(double));
	double* gpuans;
	hipMalloc((void**)&gpuans, sizeof(double) * 10000);
	hipMemcpy(gpuans, ans, sizeof(double) * 10000, hipMemcpyHostToDevice);
	calc<<<dimGrid, dimBlock >>>(gpuans);
	hipMemcpy(ans, gpuans, sizeof(double) * 10000, hipMemcpyDeviceToHost);

	double res = 0.0;
	for (int i = 0; i < 10000; i += 1) {
		res += ans[i];
	}
	stop = clock();
	double t_ns = (stop - start) / (double)(CLOCKS_PER_SEC);
	printf("%10.10f s\n", t_ns);
	printf("result is: %10.10f. \n", res);
    verification();
	return 0;
}

/* end */
