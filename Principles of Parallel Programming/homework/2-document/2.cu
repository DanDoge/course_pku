#include "hip/hip_runtime.h"
/*sigmoid function
* Author    : Huang Daoji
* StudentID : 1600017857
* Date      : 2018-04-16
*/

// header files
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>


// micros
#define M_PI_2        1.57079632679489661923	/* pi/2 */
#define M_PI_2_INV    (1.0/M_PI_2)
#define M_2_SQRTPI    1.12837916709551257390    /* 2/sqrt(pi) */
#define ERF_COEF      (1.0/M_2_SQRTPI)          // deprecated
#define threhold      1000000                   // return approximate ans if > threhold
#define SIZE          100000                    // input size


// data from here!
const char* FILE_NAME = "homework2-input";

// verification on cpu
void verification(double* input, int size) {
	double ans = 0.0;
	for (int i = 0; i < size; i += 1) {
		ans += 1 / (1 + exp(-input[i]));
	}
	printf("result on cpu: %10.10f. \n", ans);
	return;
}

// print some basic parameters, for better performance
void printDeviceProp(const hipDeviceProp_t &prop) {
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

// find a device, and quit
bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printDeviceProp(prop);
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}


// kernel function here
__global__ static void calc(double* gpuans, double* gpuinput) {
	__shared__ double tmp[100];

	int bid = blockIdx.x * 10 + blockIdx.y;
	int tid = threadIdx.x * 10 + threadIdx.y;

	int tidx = bid * 1000 + tid * 10;
	tmp[tid] = 0;
	for (int i = 0; i < 10; i += 1) {
		// base line inplementation <-- cache it!
		// <-- actually won't be faster, at least on my gpu
		double idx = gpuinput[tidx + i];
		if (idx < threhold) {
			tmp[tid] = tmp[tid] + 1 / (1 + exp(-idx));
		}
		else {
            // could it be faster?
			tmp[tid] = tmp[tid] + M_PI_2_INV * atan(M_PI_2 * (idx));
		}
	}
	// make sure all threads are done
	__syncthreads();

    // rubbish code, but works
	// now we only focuse on [0, 64)
	if (tid > 63) {
		tmp[tid - 36] = tmp[tid] + tmp[tid - 36];
	}
	__syncthreads();
	// each time we reduce the remained size by half
	// 64 -> 32 -> 16 -> ... -> 1
	int i = 32;
	while (i != 0) {
		if (tid < i) {
			tmp[tid] = tmp[tid + i] + tmp[tid];
		}
		// is everyone done...?
		__syncthreads();
		// yes! let's start the next loop!
		i /= 2;
	}
	// save ans in this block
	if (tid == 0) {
		gpuans[bid] = tmp[0];
	}
}

void read_input(double* input, int size) {
    // what if choosing mmap()?... but how to use mmap()?
	// emmmm.... won't be faster, on local machine
	FILE* fp = fopen(FILE_NAME, "r");
	if (fp) {
		for (int i = 0; i < size; i += 1) {
			fscanf(fp, "%lf\n", &input[i]);
		}
	}
	else {
		printf("error: reading input, in read_input()\n");
	}
}

int main() {
	if (!InitCUDA()) {
		return 0;
	}

	// warmup
	double* warmup = (double*)malloc(sizeof(double) * 1024 * 1024);
	double* gpuwarmup;
	hipMalloc((void**)&gpuwarmup, sizeof(double) * 1024 * 1024);
	hipMemcpy(gpuwarmup, warmup, sizeof(double) * 1024 * 1024, hipMemcpyHostToDevice);
	free(warmup);
	hipFree(gpuwarmup);

	// 100000 numbers are devided to 100 blocks, with one block having 100 threads,
	// and each therad calculate 10 numbers(and access to memory is almost aligned)
	dim3 dimBlock(10, 10);
	dim3 dimGrid(10, 10);

	// time on cpu, deprecated
	clock_t start, stop;
	start = clock();

	double* input = (double*)malloc(sizeof(double) * SIZE);
	double* gpuinput;
	hipMalloc((void**)&gpuinput, sizeof(double) * SIZE);
	read_input(input, SIZE);
	hipMemcpy(gpuinput, input, sizeof(double) * SIZE, hipMemcpyHostToDevice);

	double* ans = (double*)calloc(100, sizeof(double));
	double* gpuans;
	hipMalloc((void**)&gpuans, sizeof(double) * 100);
	hipMemcpy(gpuans, ans, sizeof(double) * 100, hipMemcpyHostToDevice);
	calc << <dimGrid, dimBlock >> >(gpuans, gpuinput);
	hipMemcpy(ans, gpuans, sizeof(double) * 100, hipMemcpyDeviceToHost);

	// the last step of reduce is done on cpu, only 100 numbers
	double res = 0.0;
	for (int i = 0; i < 100; i += 1) {
		res += ans[i];
	}

	hipFree(gpuans);
    free(ans);
	hipFree(gpuinput);
	stop = clock();
	// that's cpu time, not accurate
	double t_ns = (stop - start) / (double)(CLOCKS_PER_SEC);
	printf("%10.10f s\n", t_ns);
	printf("result is: %10.10f. \n", res);
	// am i right?
	verification(input, SIZE);
	free(input);

	// for debug on local machine
	//scanf("%ld", res);
	return 0;
}

/* end */
