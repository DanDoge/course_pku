#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#define SIZE 1024
#define bool int
#define false 0
#define true  1

int matrix[SIZE][SIZE] = {};
int result[SIZE][SIZE]    = {};

void random_generator(int** mat, int size){
    for(int i = 0; i < size; i += 1){
        for(int j = 0; j < size; j += 1){
            mat[i][j] = rand() % 1024;
        }
    }
}

bool verification(int** mat, int** res, int size){
    for(int i = 0; i < size; i += 1){
        for(int j = 0; j < size; j += 1){
            if(mat[i][j] != res[j][i]){
                return false;
            }
        }
    }
    return true;
}

bool InitCUDA(){
    int count = 0;
    hipGetDeviceCount(&count);

    if(count == 0){
        fprintf(stderr, "no device found!\n");
        return false;
    }

    for(int i = 0; i < count; i += 1){
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess){
            if(prop.major >= 1){
                break;
            }
        }
    }

    if(i == count){
        fprintf(stderr, "no device supporting CUDA!\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

__global__ static void trans(int** mat, int** res, int size){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int idx = bid * THREAD_NUM + tid;

    const int tx = idx / size;
    const int ty = idx % size;

    res[ty * size + tx] = mat[tx * size + ty];
}

int main(){
    if(!InitCUDA()){
        return 0;
    }
    random_generator(matrix, SIZE);

    int *gpumatrix, *gpuresult;
    hipMalloc((void**)&gpumatrix, sizeof(int) * SIZE * SIZE);
    hipMalloc((void**)*gpuresult, sizeof(int) * SIZE * SIZE);

    hipMemcpy(gpumatrix, matrix, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(gpuresult, result, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);

    trans<<<SIZE, SIZE, 0>>>(gpumatrix, gpuresult, SIZE);

    hipMemcpy(result, gpuresult, sizeof(int) * SIZE * SIZE, hipMemcpyDeviceToHost);

    hipFree(gpumatrix);
    hipFree(gpuresult);

    if(!verification(matrix, result, SIZE)){
        fprintf(stderr, "wrong!\n");
    }
    return 0;
}
