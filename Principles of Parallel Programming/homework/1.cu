#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>

#define bool int
#define false 0
#define true  1

void random_generator(int* mat, int size) {
	for (int i = 0; i < size; i += 1) {
		for (int j = 0; j < size; j += 1) {
			mat[i * size + j] = rand() % 1024;
		}
	}
}

bool verification(int* mat, int* res, int size) {
	for (int i = 0; i < size; i += 1) {
		for (int j = 0; j < size; j += 1) {
			if (mat[i * size + j] != res[j * size + i]) {
				return false;
			}
		}
	}
	return true;
}

void printDeviceProp(const hipDeviceProp_t &prop) {
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n"
           , prop.maxThreadsDim[0]
           , prop.maxThreadsDim[1]
           , prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n"
           , prop.maxGridSize[0]
           , prop.maxGridSize[1]
           , prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printDeviceProp(prop);
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}

__global__ static void trans(int* mat, int* res, int size, clock_t* time) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	clock_t start_time;
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		time[blockIdx.x * (size / blockDim.x) + blockIdx.y] = clock();
	}

	if(i < size && j < size){
		res[i * size + j] = mat[j * size + i];
	}

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		time[blockIdx.x * (size / blockDim.x)
             + blockIdx.y
             + size * size / (blockDim.x * blockDim.y)]
        = clock();
	}
}

int main() {
	if (!InitCUDA()) {
		return 0;
	}

	int SIZE = 0;
	scanf("%d", &SIZE);

	int* matrix = (int*)malloc(sizeof(int) * SIZE * SIZE);
	int* result = (int*)malloc(sizeof(int) * SIZE * SIZE);

	random_generator(matrix, SIZE);

	int *gpumatrix, *gpuresult;


	for (int blocksize = 1; blocksize <= 32; blocksize += 1) {
		hipMalloc((void**)&gpumatrix, sizeof(int) * SIZE * SIZE);
		hipMalloc((void**)&gpuresult, sizeof(int) * SIZE * SIZE);
		hipMemcpy(gpumatrix, matrix
                   , sizeof(int) * SIZE * SIZE
                   , hipMemcpyHostToDevice);
		int blocksum = ceil((double)SIZE / blocksize)
                       * ceil((double)SIZE / blocksize);
		dim3 dimBlock(blocksize, blocksize);
		dim3 dimGrid(ceil((double)SIZE / dimBlock.x)
                     , ceil((double)SIZE / dimBlock.y));

		clock_t* cputime = (clock_t*)malloc(sizeof(clock_t) * blocksum * 2);
		clock_t* gputime;
		hipMalloc((void**)&gputime, sizeof(clock_t) * blocksum * 2);

		trans <<<dimGrid, dimBlock, 0 >>> (gpumatrix, gpuresult, SIZE, gputime);

		hipMemcpy(result, gpuresult
                   , sizeof(int) * SIZE * SIZE
                   , hipMemcpyDeviceToHost);
		hipMemcpy(cputime, gputime
                   , sizeof(clock_t) * blocksum * 2
                   , hipMemcpyDeviceToHost);

		hipFree(gpumatrix);
		hipFree(gpuresult);
		hipFree(gputime);

		if (!verification(matrix, result, SIZE)) {
			fprintf(stderr, "wrong!\n");
		}
		else {
			fprintf(stdout, "right!\n");
		}

		clock_t min_start = cputime[0], max_end = cputime[blocksum];
		for (int i = 1; i < blocksum; i += 1) {
			if (min_start > cputime[i]) {
				min_start = cputime[i];
			}
			if (max_end < cputime[i + blocksum]) {
				max_end = cputime[i + blocksum];
			}
		}

		fprintf(stdout
                , "total time is %f, blocksize is %d\n"
                , log10((double)(max_end - min_start))
                , blocksize);
		free(cputime);
	}


	free(matrix);
	free(result);
	scanf("%d", &SIZE);
	return 0;
}
