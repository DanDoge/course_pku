#include "hip/hip_runtime.h"
/*矩阵转置作业
 * Author    : Huang Daoji
 * StudentID : 1600017857
 * Date      : 2018-04-08
 */

// header files
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

// macros
#define bool int
#define false 0
#define true  1
#define blocksize 32 // maybe the best choice for GeForce 920M


void random_generator(int* mat, int size) {
	for (int i = 0; i < size; i += 1) {
		for (int j = 0; j < size; j += 1) {
			mat[i * size + j] = rand() % 1024;
		}
	}
}

bool verification(int* mat, int* res, int size) {
	for (int i = 0; i < size; i += 1) {
		for (int j = 0; j < size; j += 1) {
			if (mat[i * size + j] != res[j * size + i]) {
				return false;
			}
		}
	}
	return true;
}

// print some basic parameters
void printDeviceProp(const hipDeviceProp_t &prop) {
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

// find a device, and quit
bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printDeviceProp(prop);
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}


// kernel function here
__global__ static void trans(int* mat, int* res, int size, clock_t* time) {
	// cache a small block
	__shared__ int cache[blocksize][blocksize];
	// locate the entry we need to trans pose
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// check the start time only if it is the first session of the block
	clock_t start_time;
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		time[blockIdx.x * (size / blockDim.x) + blockIdx.y] = clock();
	}

	// boundary check
	if(i < size && j < size){
		// cache some data,
		cache[threadIdx.y][threadIdx.x] = mat[j * size + i];
		// maybe better to do so
		__syncthreads();
		res[i * size + j] = cache[threadIdx.y][threadIdx.x];
	}

	// check the end time
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		time[blockIdx.x * (size / blockDim.x) + blockIdx.y + size * size / (blockDim.x * blockDim.y)] = clock();
	}
}

int main() {
	if (!InitCUDA()) {
		return 0;
	}

	// input the matrix size
	int SIZE = 0;
	scanf("%d", &SIZE);

	int* matrix = (int*)malloc(sizeof(int) * SIZE * SIZE);
	int* result = (int*)malloc(sizeof(int) * SIZE * SIZE);

	random_generator(matrix, SIZE);

	int *gpumatrix, *gpuresult;

	// to get run time for different blocksize, comment out for convenience.
	//for (int blocksize = 1; blocksize <= 32; blocksize += 1) {

		hipMalloc((void**)&gpumatrix, sizeof(int) * SIZE * SIZE);
		hipMalloc((void**)&gpuresult, sizeof(int) * SIZE * SIZE);
		hipMemcpy(gpumatrix, matrix, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);

		// must be ceil(), not floor(), and remember to change its type to double first.
		int blocksum = ceil((double)SIZE / blocksize) * ceil((double)SIZE / blocksize);
		dim3 dimBlock(blocksize, blocksize);
		dim3 dimGrid(ceil((double)SIZE / dimBlock.x), ceil((double)SIZE / dimBlock.y));

		clock_t* cputime = (clock_t*)malloc(sizeof(clock_t) * blocksum * 2);
		clock_t* gputime;
		hipMalloc((void**)&gputime, sizeof(clock_t) * blocksum * 2);

		trans << <dimGrid, dimBlock>> > (gpumatrix, gpuresult, SIZE, gputime);

		// get our result
		hipMemcpy(result, gpuresult, sizeof(int) * SIZE * SIZE, hipMemcpyDeviceToHost);
		hipMemcpy(cputime, gputime, sizeof(clock_t) * blocksum * 2, hipMemcpyDeviceToHost);

		// free them
		hipFree(gpumatrix);
		hipFree(gpuresult);
		hipFree(gputime);

		// am i right?
		if (!verification(matrix, result, SIZE)) {
			fprintf(stderr, "wrong!\n");
		}
		else {
			fprintf(stdout, "right!\n");
		}

		// calculate time
		clock_t min_start = cputime[0], max_end = cputime[blocksum];
		for (int i = 1; i < blocksum; i += 1) {
			if (min_start > cputime[i]) {
				min_start = cputime[i];
			}
			if (max_end < cputime[i + blocksum]) {
				max_end = cputime[i + blocksum];
			}
		}

		// "time" here actually refers to cycles
		fprintf(stdout, "total time is %f, blocksize is %d\n", log10((double)(max_end - min_start)), blocksize);
		free(cputime);
	//} // end of for(...)

	// free them!
	free(matrix);
	free(result);


	// for debug on VS2015
	//scanf("%d", &SIZE);
	return 0;
}

/* end */
